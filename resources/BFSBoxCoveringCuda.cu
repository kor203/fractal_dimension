
#include <hip/hip_runtime.h>
extern "C" {
    __device__ unsigned int firsOccurrence;

    __global__ void BFS (int threadsAmount, int firstStartNode, int networkSize, int *neighbourArray, int *neighbourStarts, int *queue, int *distances){
        int startThreadIndex = blockIdx.x * blockDim.x + threadIdx.x;

        if (startThreadIndex < threadsAmount){
            int startNodeIndex = startThreadIndex + firstStartNode;
            int threadOffset = startThreadIndex * networkSize;
            int queueNextElement = threadOffset;
            int queueLastElement = threadOffset;
            for (int i = 0; i < networkSize; i++)
                distances[i + threadOffset] = INT_MAX;
            if (startNodeIndex < networkSize)
                distances[startNodeIndex + threadOffset] = 0;
            queue[queueNextElement] = startNodeIndex;
            queueLastElement++;

            while (queueNextElement != queueLastElement){
                for (int connectionIndex = neighbourStarts[queue[queueNextElement]]; connectionIndex < neighbourStarts[queue[queueNextElement] + 1]; connectionIndex++){
                    int neighbourIndex = neighbourArray[connectionIndex];
                    if (distances[neighbourIndex + threadOffset] == INT_MAX){
                        distances[neighbourIndex + threadOffset] = distances[queue[queueNextElement] + threadOffset] + 1;
                        queue[queueLastElement] = neighbourIndex;
                        queueLastElement++;
                    }
                }
                queueNextElement++;
            }
        }
    }

    __global__ void findUsedBoxes (int startNodeIndex, int startThreadIndex, int boxSize, int minBoxSize, int networkSize, int *distances, int *boxCovering, int *blockedBoxes){
        int threadID = blockIdx.x * blockDim.x + threadIdx.x;

        if (threadID < startNodeIndex)
            if (distances[threadID + startThreadIndex * networkSize] >= boxSize)
                blockedBoxes[boxCovering[threadID + (boxSize - minBoxSize) * networkSize]] = 1;
            //else
            //    blockedBoxes[boxCovering[threadID + (boxSize - minBoxSize) * networkSize]] = 0;
    }

    __global__ void findFirstFreeBox (int networkSize, int *blockedBoxes){
        /*int iterator = 0;
        while (blockedBoxes[iterator] == 1)
            iterator++;
        firsOccurrence = iterator;*/

        int threadID = blockIdx.x * blockDim.x + threadIdx.x;

        if (threadID < networkSize){
            if (threadID == 0){
                firsOccurrence = networkSize;
                __threadfence();
            }
            if (blockedBoxes[threadID] == 0)
                atomicMin(&firsOccurrence, threadID);
        }
    }

    __global__ void resetBlocked (int networkSize, int *blockedBoxes){
        int threadID = blockIdx.x * blockDim.x + threadIdx.x;

        blockedBoxes[threadID] = 0;
    }

    __global__ void setBox (int startNodeIndex, int boxSizeIndex, int networkSize, int *boxCovering){
        boxCovering[startNodeIndex + boxSizeIndex * networkSize] = firsOccurrence;
    }
}